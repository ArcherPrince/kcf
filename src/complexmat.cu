#include "hip/hip_runtime.h"
#include "complexmat.hpp"

__global__ void sqr_norm_kernel(int n, float *out, const float *data, float rows, float cols)
{
    extern __shared__ float sdata[];
    int i = blockDim.x * threadIdx.y + threadIdx.x;
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = 2 * (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

    sdata[i] = 0;
    sdata[i] = data[threadId] * data[threadId] + data[threadId + 1] * data[threadId + 1];
    __syncthreads();

    for (unsigned int s = (blockDim.x * blockDim.y + 1) / 2, old_s = blockDim.x * blockDim.y; s > 0; s >>= 1) {

        if (old_s & 1) s += 1;

        if (i < s && i + s < old_s) {
            sdata[i] += sdata[i + s];
        }
        old_s = s;
        __syncthreads();
    }

    if (i == 0) {
        atomicAdd(&out[blockId / n], sdata[0] / (rows * cols));
    }
}

void ComplexMat::sqr_norm(DynMem &result) const
{
    CudaSafeCall(hipMemsetAsync(result.deviceMem(), 0, n_scales * sizeof(float)));

    dim3 threadsPerBlock(rows, cols);
    dim3 numBlocks(n_channels / n_scales, n_scales);

    sqr_norm_kernel<<<numBlocks, threadsPerBlock, rows * cols * sizeof(float)>>>(
        n_channels / n_scales, result.deviceMem(), (float*)this->p_data.deviceMem(), rows, cols);
    CudaCheckError();

    return;
}

__global__ void sqr_mag_kernel(const float *data, float *result)
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = 2 * (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

    result[threadId] = data[threadId] * data[threadId] + data[threadId + 1] * data[threadId + 1];
    result[threadId + 1] = 0;
}

ComplexMat ComplexMat::sqr_mag() const
{
    ComplexMat result(this->rows, this->cols, this->channels(), this->n_scales);

    dim3 threadsPerBlock(rows, cols);
    dim3 numBlocks(n_channels / n_scales, n_scales);
    sqr_mag_kernel<<<numBlocks, threadsPerBlock, 0>>>((float*)this->p_data.deviceMem(), (float*)result.p_data.deviceMem());
    CudaCheckError();

    return result;
}

__global__ void conj_kernel(const float *data, float *result)
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = 2 * (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

    result[threadId] = data[threadId];
    result[threadId + 1] = -data[threadId + 1];
}

ComplexMat ComplexMat::conj() const
{
    ComplexMat result(this->rows, this->cols, this->channels(), this->n_scales);

    dim3 threadsPerBlock(rows, cols);
    dim3 numBlocks(n_channels / n_scales, n_scales);
    conj_kernel<<<numBlocks, threadsPerBlock, 0>>>((float*)this->p_data.deviceMem(), (float*)result.p_data.deviceMem());
    CudaCheckError();

    return result;
}

__global__ static void sum_channels(float *dest, const float *src, uint channels, uint num_channel_elem)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= num_channel_elem)
        return;

    float acc = 0;
    for (uint i = 0; i < channels; ++i)
        acc += src[idx + i * num_channel_elem];
    dest[idx] = acc;
}

ComplexMat ComplexMat::sum_over_channels() const
{
    assert(p_data.num_elem == n_channels * rows * cols);

    uint n_channels_per_scale = n_channels / n_scales;
    uint scale_offset = n_channels_per_scale * rows * cols;

    ComplexMat_ result(this->rows, this->cols, 1, n_scales);

    const uint total = rows * cols * 2;
    const dim3 threads(256);
    const dim3 blocks((total + threads.x - 1) / threads.x);

    for (uint scale = 0; scale < n_scales; ++scale) {
        sum_channels<<<blocks, threads>>>(reinterpret_cast<float*>(result.p_data.deviceMem() + scale * scale_offset),
                                          reinterpret_cast<const float*>(p_data.deviceMem() + scale * scale_offset),
                                          n_channels_per_scale, total);
    }
    CudaSafeCall(hipStreamSynchronize(hipStreamPerThread));
    return result;
}

__global__ void same_num_channels_mul_kernel(const float *data_l, const float *data_r, float *result)
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = 2 * (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

    result[threadId] = data_l[threadId] * data_r[threadId] - data_l[threadId + 1] * data_r[threadId + 1];
    result[threadId + 1] = data_l[threadId] * data_r[threadId + 1] + data_l[threadId + 1] * data_r[threadId];
}

// element-wise per channel multiplication, division and addition
ComplexMat ComplexMat::operator*(const ComplexMat &rhs) const
{
    assert(rhs.n_channels == n_channels && rhs.cols == cols && rhs.rows == rows);

    ComplexMat result(this->rows, this->cols, this->channels(), this->n_scales);

    dim3 threadsPerBlock(rows, cols);
    dim3 numBlocks(n_channels / n_scales, n_scales);
    same_num_channels_mul_kernel<<<numBlocks, threadsPerBlock, 0>>>((float*)this->p_data.deviceMem(),
                                                                    (float*)rhs.p_data.deviceMem(),
                                                                    (float*)result.p_data.deviceMem());
    CudaCheckError();

    return result;
}

__global__ void same_num_channels_div_kernel(const float *data_l, const float *data_r, float *result)
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = 2 * (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

    result[threadId] = (data_l[threadId] * data_r[threadId] + data_l[threadId + 1] * data_r[threadId + 1]) /
                       (data_r[threadId] * data_r[threadId] + data_r[threadId + 1] * data_r[threadId + 1]);
    result[threadId + 1] = (data_l[threadId + 1] * data_r[threadId] - data_l[threadId] * data_r[threadId + 1]) /
                           (data_r[threadId] * data_r[threadId] + data_r[threadId + 1] * data_r[threadId + 1]);
}

ComplexMat ComplexMat::operator/(const ComplexMat &rhs) const
{
    assert(rhs.n_channels == n_channels && rhs.cols == cols && rhs.rows == rows);

    ComplexMat result(this->rows, this->cols, this->channels(), this->n_scales);

    dim3 threadsPerBlock(rows, cols);
    dim3 numBlocks(n_channels / n_scales, n_scales);
    same_num_channels_div_kernel<<<numBlocks, threadsPerBlock, 0>>>((float*)this->p_data.deviceMem(),
                                                                    (float*)rhs.p_data.deviceMem(),
                                                                    (float*)result.p_data.deviceMem());
    CudaCheckError();

    return result;
}

__global__ void same_num_channels_add_kernel(const float *data_l, const float *data_r, float *result)
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = 2 * (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

    result[threadId] = data_l[threadId] + data_r[threadId];
    result[threadId + 1] = data_l[threadId + 1] + data_r[threadId + 1];
}

ComplexMat ComplexMat::operator+(const ComplexMat &rhs) const
{
    assert(rhs.n_channels == n_channels && rhs.cols == cols && rhs.rows == rows);

    ComplexMat result(this->rows, this->cols, this->channels(), this->n_scales);

    dim3 threadsPerBlock(rows, cols);
    dim3 numBlocks(n_channels / n_scales, n_scales);
    same_num_channels_add_kernel<<<numBlocks, threadsPerBlock, 0>>>((float*)this->p_data.deviceMem(),
                                                                    (float*)rhs.p_data.deviceMem(),
                                                                    (float*)result.p_data.deviceMem());
    CudaCheckError();

    return result;
}

__global__ void constant_mul_kernel(const float *data_l, float constant, float *result)
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = 2 * (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

    result[threadId] = data_l[threadId] * constant;
    result[threadId + 1] = data_l[threadId + 1] * constant;
}

ComplexMat ComplexMat::operator*(const float &rhs) const
{
    ComplexMat result(this->rows, this->cols, this->channels(), this->n_scales);

    dim3 threadsPerBlock(rows, cols);
    dim3 numBlocks(n_channels / n_scales, n_scales);
    constant_mul_kernel<<<numBlocks, threadsPerBlock, 0>>>((float*)this->p_data.deviceMem(),
                                                           rhs,
                                                           (float*)result.p_data.deviceMem());
    CudaCheckError();

    return result;
}

__global__ void constant_add_kernel(const float *data_l, float constant, float *result)
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = 2 * (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

    result[threadId] = data_l[threadId] + constant;
    result[threadId + 1] = data_l[threadId + 1];
}

ComplexMat ComplexMat::operator+(const float &rhs) const
{
    ComplexMat result(this->rows, this->cols, this->channels(), this->n_scales);

    dim3 threadsPerBlock(rows, cols);
    dim3 numBlocks(n_channels / n_scales, n_scales);
    constant_add_kernel<<<numBlocks, threadsPerBlock, 0>>>((float*)this->p_data.deviceMem(),
                                                           rhs,
                                                           (float*)result.p_data.deviceMem());
    CudaCheckError();

    return result;
}

__global__ void one_channel_mul_kernel(const float *data_l, const float *data_r, float *result)
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = 2 * (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);
    int one_ch_index = 2 * ((threadIdx.y * blockDim.x) + threadIdx.x);

    result[threadId] = data_l[threadId] * data_r[one_ch_index] - data_l[threadId + 1] * data_r[one_ch_index + 1];
    result[threadId + 1] = data_l[threadId] * data_r[one_ch_index + 1] + data_l[threadId + 1] * data_r[one_ch_index];
}

// multiplying element-wise multichannel by one channel mats (rhs mat is with one channel)
ComplexMat ComplexMat::mul(const ComplexMat &rhs) const
{
    assert(rhs.n_channels == 1 && rhs.cols == cols && rhs.rows == rows);

    ComplexMat result(this->rows, this->cols, this->channels(), this->n_scales);

    dim3 threadsPerBlock(rows, cols);
    dim3 numBlocks(n_channels / n_scales, n_scales);
    one_channel_mul_kernel<<<numBlocks, threadsPerBlock, 0>>>((float*)this->p_data.deviceMem(),
                                                              (float*)rhs.p_data.deviceMem(),
                                                              (float*)result.p_data.deviceMem());
    CudaCheckError();

    return result;
}

__global__ void scales_channel_mul_kernel(float *data_l, float *data_r, float *result)
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = 2 * (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);
    int one_ch_index = 2 * ((threadIdx.y * blockDim.x) + threadIdx.x + blockIdx.x * blockDim.x * blockDim.y);

    result[threadId] = data_l[threadId] * data_r[one_ch_index] - data_l[threadId + 1] * data_r[one_ch_index + 1];
    result[threadId + 1] = data_l[threadId] * data_r[one_ch_index + 1] + data_l[threadId + 1] * data_r[one_ch_index];
}

// multiplying element-wise multichannel by one channel mats (rhs mat is with multiple channel)
// ComplexMat ComplexMat::mul2(const ComplexMat &rhs) const
// {
//     assert(rhs.n_channels == n_channels / n_scales && rhs.cols == cols && rhs.rows == rows);

//     ComplexMat result(this->rows, this->cols, this->channels(), this->n_scales);

//     dim3 threadsPerBlock(rows, cols);
//     dim3 numBlocks(n_channels / n_scales, n_scales);
//     scales_channel_mul_kernel<<<numBlocks, threadsPerBlock, 0>>>(this->p_data, rhs.p_data, result.p_data);
//     CudaCheckError();

//     return result;
// }

// void ComplexMat::operator=(ComplexMat &&rhs)
// {
//     cols = rhs.cols;
//     rows = rhs.rows;
//     n_channels = rhs.n_channels;
//     n_scales = rhs.n_scales;

//     p_data = rhs.p_data;

//     rhs.p_data = nullptr;
// }
